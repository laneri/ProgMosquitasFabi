#include "hip/hip_runtime.h"
#include <Random123/philox.h> // philox headers
#include <Random123/u01.h>    // to get uniform deviates [0,1]
typedef r123::Philox2x32 RNG; // particular counter-based RNG

#include <cstdlib>
#include <iostream>
#include <fstream>
#include <vector>
#include "ran2.h"
#include <cmath>
#include <thrust/device_vector.h>
#include "gpu_timer.h"
#include "parametros.h"

int tiempo_entre_oviposiciones(int dia){
	int t;
	//defino tiempos de oviposiciรณn y maduraciรณn en funciรณn de la temperatura	
	if(dia >= 1 && dia < 80) t=tovip3;      //<T>=18 // acรก es necesario definirlo tomando el extremo
	if(dia >= 80 && dia <= 140)t=tovip2b;   //<T>=23
  	if(dia > 140 || dia < 260) t=tovip1;    //<T>=30
 	if(dia >= 260 && dia <= 320)t=tovip2a;  //<T>=27
 	if(dia > 320) t=tovip3;                 //<T>=18

	return t;}


__global__ void kernel_reproducir(int *estado, int *edad, int *tacho,int *TdV, int *pupacion,int *manzana, int *N_mobil, int dia, int tovip, int *nacidos)
{
    	int indice=N_mobil[0];
	    int id = blockIdx.x*blockDim.x + threadIdx.x;

  		if(id < indice)
  		{
  		RNG philox;         
	    RNG::ctr_type c={{}};
	    RNG::key_type k={{}};
	    RNG::ctr_type r;
	    k[0]=id; 
	    c[1]=dia;
	    c[0]=SEMILLAGLOBAL; 
		
    	r = philox(c, k); 
     	double azar=(u01_closed_closed_32_53(r[0]));
/*Si la mosquita esta viva, esta en edad adulta y en el tiempo de oviposicion entonces*/

			if(estado[id] == ESTADOVIVO && edad[id] > pupacion[id] && edad[id]%tovip == 0){
/*Si el tacho en el que nacio tiene lugar entonces*/
				/*Antes estaba asi y andaba...*/
				int tach=tacho[id];     //tach es un entero que me indica el numero de tacho en el que esta cada mosquita
				
				 int iovip=1;
				 //10 + (azar*25); //iovip es el numero de huevos que pone cada mosquita
						
				atomicAdd(nacidos+tach,iovip); /*sumo iovip HUEVOS en la posicion del vector nacidos (que tiene NTACHOS elementos)
				nacidos[0+tach] en el puntero al primer elemento del vector nacidos desplazado en tach elementos
				el vector nacidos tiene el numero de nacidos en cada tacho Ej: nacidos[0]=numero de nacidos en el tacho 1*/
				
				
		    }//cierro loop para mosquitasvivas y  maduras
  		}//cierro loop de hilos    
};

//mortalidades varias	
__global__ void matar_kernel(int *estado, int *edad, int *tacho,int *pupacion, int *TdV,int *N_mobil, int dia)
{
	int N=N_mobil[0];
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	
	if(id<N){		
	 	RNG philox;         
	    RNG::ctr_type c={{}};
	    RNG::key_type k={{}};
	    RNG::ctr_type r;
	    k[0]=id; 
	    c[1]=dia;
	    c[0]=SEMILLAGLOBAL; 
		
    	r = philox(c, k); 
     	double azar=(u01_closed_closed_32_53(r[0]));
     	//acá incluye mortalidad de huevos,pupas y adultos con cierta probabilidad, y además muertes por vejez
		    if (estado[id]==ESTADOVIVO && edad[id] < pupacion[id]){if(azar < moracu)estado[id]=ESTADOMUERTO;}
		    if (estado[id]==ESTADOVIVO && edad[id] == pupacion[id]){if(azar < morpupad)estado[id]=ESTADOMUERTO;}
		    if (estado[id]==ESTADOVIVO && edad[id] > pupacion[id]){if(azar < morad)estado[id]=ESTADOMUERTO;}
		    if (estado[id]==ESTADOVIVO && edad[id] >= TdV[id])estado[id]=ESTADOMUERTO;
	}
};

//muerte de las mosquitas por vejez
/*__global__ void matar_viejos_kernel(int *estado, int *edad, int *tacho, int *TdV,int *N_mobil,int dia)
{

	    int N=N_mobil[0];
	    int id = blockIdx.x*blockDim.x + threadIdx.x;

	    if(id<N){                     
	        if(estado[id]==ESTADOVIVO && edad[id] >= TdV[id])estado[id]=ESTADOMUERTO;
	    }
};
*/
__global__ void descacharrado_kernel(int *estado, int *edad, int *tacho, int *pupacion,int *N_mobil,int dia, int ntach)
{
    	int N=N_mobil[0];
	    int id = blockIdx.x*blockDim.x + threadIdx.x;

	    if(id<N){
	    if (estado[id] == ESTADOVIVO && edad[id] < pupacion[id] && tacho[id] == ntach)estado[id]=ESTADOMUERTO;
    	}
};

__global__ void envejecer_kernel(int *estado, int *edad,int *pupacion,int *N_mobil,int dia)
{
    	int N=N_mobil[0];
	    int id = blockIdx.x*blockDim.x + threadIdx.x;

        if(id<N){
            if(dia < 80 || dia > 320){
                if(estado[id] == ESTADOVIVO && edad[id] > pupacion[id])edad[id]++;} //ADULTAS
  		    else{
  		        if (estado[id]== ESTADOVIVO)edad[id]++;}
    	}
};

//es muy lento
/*__global__ void conteo_kernel(int *estado, int *edad,int *tacho,int *pupacion, int *tach,int *N_mobil)
{
       	int N=N_mobil[0];
	    int id = blockIdx.x*blockDim.x + threadIdx.x;
	    
		if(id<N){
			if(edad[id] < pupacion[id] && estado[id] == ESTADOVIVO){ 
    			int j=tacho[id]; 
	    		tach[j]++;
			}
		} 
};
*/


// otro functorcito usado para las estadisticas desagregadas
struct acuaticoeneltacho{
	int m;
    int t;
	acuaticoeneltacho(int m_, int t_):m(m_),t(t_){};
    
	__device__ bool operator()(thrust::tuple<int,int> tupla)
	{
        int tach=thrust::get<0>(tupla);
        int edad=thrust::get<1>(tupla);
		return (tach==m && edad<t);
	}
};

// functorcito para contar adultos en la población
struct poblacion_1{
	__device__ bool operator()(thrust::tuple<int,int> tupla)
	{
        int pupacion=thrust::get<0>(tupla);
        int edad=thrust::get<1>(tupla);
		return (edad >= pupacion);
	}
};

// functorcito para contar acuáticos en la población
struct poblacion_2{
	__device__ bool operator()(thrust::tuple<int,int> tupla)
	{
        int pupacion=thrust::get<0>(tupla);
        int edad=thrust::get<1>(tupla);
		return (edad < pupacion);
	}
};

/////////////////////////////////////////////////////////////////////////////////////////
// Clase bichos: toda la info sobre todos los bichos, y sus funciones
struct bichos{

	thrust::device_vector<int> estado; // Vivo o Muerto 0/1 
	thrust::device_vector<int> edad; // tiene num de mosqu elementos y los valores van de 0 a MAXIMAEDAD   
	thrust::device_vector<int> tacho; // numero de tacho en que se encuentra cada mosquita valores=0 a NUMEROTACHOS   
	thrust::device_vector<int> TdV;  //tiempo de vida de cada mosquita
	thrust::device_vector<int> pupacion; //dia de paso de pupa a adulta de cada mosquita
	thrust::device_vector<int> manzana; //numero de manzana de cada mosquita
	//thrust::device_vector<int> tach; //numero de tacho de cada mosquita
	thrust::device_vector<int> nacidos; // tiene el num de tachos elementos, numero de nacidos por tacho

	thrust::device_vector<int> N_mobil; // Numero de bichos fluctuante (1 elemento)

	// punteros crudos a los arrays para pasarselos a kernels
	int *raw_edad;
	int *raw_tacho;
	int *raw_estado;
	int *raw_TdV;
	//int *raw_tach;
	int *raw_pupacion;
	int *raw_N_mobil;
	int *raw_manzana;
	int *raw_nacidos;
	
	//constructor	
	bichos(int N_){
	// alocamos el maximo posible
	estado.resize(MAXIMONUMEROBICHOS);	
	tacho.resize(MAXIMONUMEROBICHOS);	
	edad.resize(MAXIMONUMEROBICHOS);
	pupacion.resize(MAXIMONUMEROBICHOS);
	TdV.resize(MAXIMONUMEROBICHOS);
	manzana.resize(MAXIMONUMEROBICHOS);
	//tach.resize(MAXIMONUMEROBICHOS);

	N_mobil.resize(1);

	// nacidos en cada tacho, inicialmente 0
	nacidos.resize(NUMEROTACHOS);
	thrust::fill(nacidos.begin(),nacidos.end(),0);

	thrust::fill(estado.begin(),estado.end(),0);
	thrust::fill(edad.begin(),edad.end(),0);
	thrust::fill(tacho.begin(),tacho.end(),0);
	thrust::fill(pupacion.begin(),pupacion.end(),0);
	thrust::fill(TdV.begin(),TdV.end(),0);
	thrust::fill(manzana.begin(),manzana.end(),0);
	
	// inicializacion raw pointers
	raw_edad=thrust::raw_pointer_cast(edad.data());
	raw_tacho=thrust::raw_pointer_cast(tacho.data());
	raw_estado=thrust::raw_pointer_cast(estado.data());
	raw_TdV=thrust::raw_pointer_cast(TdV.data());
	raw_manzana=thrust::raw_pointer_cast(manzana.data());
	raw_N_mobil=thrust::raw_pointer_cast(N_mobil.data());
	//raw_tach=thrust::raw_pointer_cast(tach.data());
	raw_pupacion=thrust::raw_pointer_cast(pupacion.data());
	raw_nacidos=thrust::raw_pointer_cast(nacidos.data());

    //std::cout<<"VoM\ttacho\tedad\tTdV\ttpupad\tmanzana" << std::endl;
	/*condiciones iniciales donde N sera el numero de bichos*/
 	for(int i=0;i < N_;i++){
		estado[i]=ESTADOVIVO; 		    //todos vivos inicialmente
		tacho[i]=i;				        //tacho en el que se encuentra la mosquita
		edad[i]=ran2(&semilla)*7+19; 	//edad 
		pupacion[i]=tpupad-2+(ran2(&semilla)*5);//dia de pupacion (entre los 15 y 19 dias)
		TdV[i]=ran2(&semilla)*6+27 ;	//tiempo de vida de 27 a 32
		manzana[i]=(int) (i/5);         //manzana en la que se encuentra
		
	//	std::cout << estado[i] << "\t" << tacho[i] << "\t" << edad[i] << "\t" << TdV[i] << "\t" << pupacion[i] << "\t" << manzana[i] << "\n";
	 //std::cout << "pupacion" << pupacion[i]<< std::endl;
}

std::cout << "inicializacion lista" << std::endl;

	N_mobil[0]=N_;
	};	

	
	void mortalidades(int dia){

	int N=N_mobil[0];
	
	    matar_kernel<<<(N+256-1)/256,256>>>(raw_estado,raw_edad,raw_tacho,raw_pupacion,raw_TdV,raw_N_mobil, dia);	
		hipDeviceSynchronize();
	};
	
	//descachrarrado
	void descacharrado(int dia,int descach){
	int N=N_mobil[0];	
		if(dia%7 == 0 && dia > 120 && dia < 320){
  			for(int itach=0;itach < descach;itach++){
    			int ntach=ran2(&semilla)*NUMEROTACHOS;
  				descacharrado_kernel<<<(N+256-1)/256,256>>>(raw_estado, raw_edad, raw_tacho, raw_pupacion, raw_N_mobil,dia,ntach);
			}    	
   		}
	};

	//conteo de huevos
	void conteo_huevos(int dia){
	int N=N_mobil[0];
//		thrust::fill(tach.begin(),tach.begin() + N,0);
	    //demora demasiado
		//conteo_kernel<<<(N+256-1)/256,256>>>(raw_estado,raw_edad,raw_tacho,raw_pupacion,raw_N_mobil);
		//hipDeviceSynchronize();
		
 	/* 	for(int i=0;i < N; i++){ 
			if(edad[i] < pupacion[i] && estado[i] == ESTADOVIVO){ 
    			int j=tacho[i]; 
	    		tach[j]++;
			}
		} 
 */
	};

    //nacimientos
	void reproducir(int dia,int tovip){
	    
	int indice=N_mobil[0];
	if(indice==0) {
		std::cout << "NO HAY MAS MOSQUITAS PARA REPRODUCIRSE" << std::endl; 	
	exit(1);
	}else{
	//nacimientos
	int mosqsat=0;
	std::cout << "antes kernel reproducir " << std::endl;
	//reinicializo en cero los nacidos en el paso anterior que ahora ya no son mas nacidos porque crecieron 
	thrust::fill(nacidos.begin(),nacidos.end(),0);
	// reproduce, calculando nacidos por tacho antes
	kernel_reproducir<<<(indice+256-1)/256,256>>>(raw_estado,raw_edad,raw_tacho,raw_TdV,raw_pupacion,raw_manzana,raw_N_mobil,dia,tovip,raw_nacidos);
	hipDeviceSynchronize();
	std::cout << "despues kernel reproducir " << std::endl; 
		// agrego todos los nacidos al final del array original, tacho a tacho
		int index=indice;
        bool nosaturo=1;
		for(int m=0;m<NUMEROTACHOS;m++){
			//std::cout << "listo nacidos " << std::endl; 
 
            //antiguos=thrust::count_if(tacho.begin(),tacho.begin()+N,iguala(m));
//calculo el nunmero de acuaticos en cada tacho
            int antiguos=thrust::count_if(
                thrust::make_zip_iterator(thrust::make_tuple(tacho.begin(),edad.begin())),
                thrust::make_zip_iterator(thrust::make_tuple(tacho.begin()+indice,edad.begin()+indice)),
                acuaticoeneltacho(m,tpupad)
			);
			std::cout << "Acuaticos en Tacho " << antiguos <<std::endl;
          	//los nuevos vienen del kernel reproducir  
            int nuevos=nacidos[m];
			std::cout << "NUEVOS NACIDOS " << nuevos <<std::endl; 
			//no saturo me asegura que no me excedo del tamaño del array
            nosaturo=((index+nuevos) < MAXIMONUMEROBICHOS);
            if((antiguos+nuevos) < SAT && nosaturo==1){ 						//cond. saturacion 
            thrust::fill(estado.begin()+index,estado.begin()+index+nuevos,0);	//todos vivos(0)	
			thrust::fill(edad.begin()+index,edad.begin()+index+nuevos,0);		//todos nacen con edad 0	
			thrust::fill(tacho.begin()+index,tacho.begin()+index+nuevos,m); 	//nacen en el tacho m
			thrust::fill(TdV.begin()+index,TdV.begin()+index+nuevos,rand()%3+28); // con tiempo de vida 28 a 30
			index+=nuevos;		//actualizo el indice para me marque siempre en la ultima mosquita que nacio 
			}else if((antiguos+nuevos) > SAT){
				std::cout << "Nos pasamos del maximo por tacho" <<std::endl;
			}else{std::cout << "Nos pasamos del maximo de bichos totales" << nuevos <<std::endl;}
		}
	// problema si esto satura saturan los tachos mas grandes, ver como cambiar esto por ejemplo llnando tachos al azar
        if(nosaturo==0) std::cout << "algunos tachos no se rellenaron por saturacion del total de mosquitas" << std::endl;
	//imprimo el indice
	std::cout << "index " << index <<std::endl; 
	
	// actualiza el indice movil hasta el ultimo bicho vivo
		if(index<MAXIMONUMEROBICHOS) {
			N_mobil[0]=index;
		}
		// caso contrario satura al maximo
		else{
			std::cout << "Demasiados Bichos!" << std::endl;
			 N_mobil[0]=MAXIMONUMEROBICHOS-1;	
			}	
	}	
	
	/* 	for(int i=0;i < indice;i++){
			if(estado[i] == ESTADOVIVO && edad[i] > pupacion[i] && edad[i]%tovip == 0){
				if (tach[tacho[i]] < sat){
					  int iovip=10 + (ran2(&semilla)*25); 
   						for(int ik=0;ik < iovip;ik++){ 
 						estado[indice]=ESTADOVIVO;
 						edad[indice]=1;   
 						tacho[indice]=tacho[i]; 
		         		pupacion[indice]=tpupad-2+(ran2(&semilla)*5);	//dias de pupacion
	         			TdV[indice]=ran2(&semilla)*6+27;  
						manzana[indice]=manzana[i];
						int j=tacho[indice];
 						tach[j]++;
						indice++;
   						} 
				}//cierro tach
				else{	
			        mosqsat++;   			//sumo las mosquitas que no pudieron poner en este tiempo (solo como dato)
				         for(int j=0;j < ntachito;j++){      //si no tiene lugar en su tacho migra a otro 
			          		if(tach[j] < sat){   	     // se fija si sus huevos van a tener lugar 
			           		tacho[i]=j;          	     //se mueve
						    int iovip=10 + (ran2(&semilla)*25); 
   							for(int ik=0;ik < iovip;ik++){ 
 							estado[indice]=ESTADOVIVO;
 							edad[indice]=1;   
 							tacho[indice]=tacho[i]; 
		         			pupacion[indice]=tpupad-2+(ran2(&semilla)*5);	//dias de pupacion
	         				TdV[indice]=ran2(&semilla)*6+27;  
							int j=tacho[indice];
 							tach[j]++;
							indice++;
   							}
						j=ntachito;
						}
					}
				} 				
   			} 
		} */
//		actualiza el numero de bichos si no se sobrepasa el maximo
		//N_mobil[0]=indice;

	};

    //Recalcular -> eliminar muertos y dejar vivos
    void recalcularN(){

		auto zip_iterator=
		thrust::make_zip_iterator(thrust::make_tuple(edad.begin(),tacho.begin(),pupacion.begin(),TdV.begin(),manzana.begin()));
		// ordenamos segun estado 0-vivo, 1-muerto
		int N=N_mobil[0];
		thrust::sort_by_key(estado.begin(), estado.begin() + N,zip_iterator);		
		//std::cout << "N en recalcular N " << N <<std::endl; 
		// y ahora determinamos la posicion del primer muerto = N_mobil
		auto iter=thrust::find(estado.begin(),estado.begin() + N, ESTADOMUERTO);
		
		N_mobil[0]= iter-estado.begin();//me da la longitud del vector
		std::cout << "N_mobil " << N_mobil[0] <<std::endl;
	};
	
	// Numero de mosquitas vivas
	int vivos(int dia){

	int N=N_mobil[0];

    int poblacion = thrust::count(estado.begin(), estado.begin() + N, ESTADOVIVO);
	return poblacion;
	};

    //población de acuáticos
	int acuaticos(int dia){

	int N=N_mobil[0];
	
    int ac=thrust::count_if(
                thrust::make_zip_iterator(thrust::make_tuple(edad.begin(),pupacion.begin())),
                thrust::make_zip_iterator(thrust::make_tuple(edad.begin() +  N,pupacion.begin() + N)),
                poblacion_1()
            );

	return ac;
	};

    //población de adultos
	int adultos(int dia){

	int N=N_mobil[0];
	    int ad=thrust::count_if(
                thrust::make_zip_iterator(thrust::make_tuple(edad.begin(),pupacion.begin())),
                thrust::make_zip_iterator(thrust::make_tuple(edad.begin() +  N,pupacion.begin() + N)),
                poblacion_2()
            );

		return ad;
	};

	//envejecer población

   	void envejecer(int dia){
	int N=N_mobil[0];
        envejecer_kernel<<<(N + 256-1)/256,256>>>(raw_estado,raw_edad,raw_pupacion,raw_N_mobil,dia);
	}; 

	// Envejezco a toda la población un día (aumento en 1): recorre los bichos y les aumenta la edad en una unidad
	void envejecerthrust(){
	int N=N_mobil[0];
	using namespace thrust::placeholders;
	//envejecen las que estan vivas por eso hasta N 	
	thrust::transform(edad.begin(),edad.begin()+N, edad.begin(),_1+1);
	
	};
};


int main(){

 	std::ofstream outfile, outfile1, outfile2;
   	outfile.open("Poblacion_total_GPU.dat");
   	outfile1.open("Poblacion_adultos_GPU.dat");
    outfile2.open("Poblacion_acuaticos_GPU.dat");

	int descach=round(NUMEROTACHOS*prop);//cantidad de tachos que vacรญo con la propaganda 
	
	gpu_timer Reloj_GPU;
	Reloj_GPU.tic();
    
    bichos mosquitas(Ninicial);

	for(int dia = 1; dia <= Ndias; dia++){
	int tovip=tiempo_entre_oviposiciones(dia);
	
	std::cout << "reproducir" << std::endl;
	mosquitas.reproducir(dia,tovip);
	
	std::cout << "matar" << std::endl;
	mosquitas.mortalidades(dia);//fusione muerte x vejez con mortalidades varias en un solo kernel
	//mosquitas.muerte_x_vejez(dia);
	
	std::cout << "descacharrar" << std::endl;
	mosquitas.descacharrado(dia,descach); 
	
	//Esto no se para que sirve
	//mosquitas.conteo_huevos(dia);
	std::cout << "envejezco poblacion" << std::endl;
	//mosquitas.envejecer(dia);
	mosquitas.envejecerthrust();
	
	std::cout << "recalculo indice de mosquitas vivas" << std::endl;
	mosquitas.recalcularN(); 

	std::cout << "DIA" << dia << std::endl;

	int vivas=mosquitas.vivos(dia);
	int adultos=mosquitas.adultos(dia);
	int acuaticos=mosquitas.acuaticos(dia);
	outfile << dia << "\t" << vivas << std::endl;
	outfile1 << dia << "\t" << adultos << std::endl;
	outfile2 << dia << "\t" << acuaticos << std::endl;
	 

	}

/* 	void avanza_dia(int dia)
    {
        std::cout << "reproducir" << std::endl;
		reproducir(dia); 		//nacimientos

        std::cout << "matar" << std::endl;
        matar(dia);			//mortalidades varias

        matar_viejos(dia);		//mortalidad por vejez
		descacharrar_tacho(dia);	//descacharrar tacho
		envejecer();			//envejecer un dia la poblacion
		recalcularN();
	};
 */
	

    double t=Reloj_GPU.tac()/60000; //de milisegundos -> minutos
    printf("Tiempo en GPU: %lf minutos\n",t);

return 0;

//cierro archivos
outfile.close();
outfile1.close();
outfile2.close();
}// end for main

