#include "hip/hip_runtime.h"

//*************************************************************************************************************
//                                  Programa FORTRAN de mosquita para una manzana versión (13/6/2020) 
//                                  Autora Fabiana Laguna
//*************************************************************************************************************
/*En la naturaleza, cada oviposicion son aprox. 64 huevos y la mitad son hembras. En este código solamente se modela la dinámica de las hembras. Si se quisiera agregar a los machos, se multiplica por dos. Para ello se considera:

1- el número de huevos que deposita la hembra  entre 10 y 35 por oviposicion (distribució uniforme).
2- la mortalidad diaria de huevos, pupas, larvas y adultas independiente de la Temperatura.
3- el 83% de las larvas pasan a adultas jovenes, es decir, mueren con probab 0.17 al pasar del agua al aire
4- el nro. de oviposiciones por hembra depende de la temperatura: 
        - 0 ovip. a 18 grados (es decir, que en invierno las hembras no ponen huevos)
        - 3 o 4 ovip. a 25 grados 
        - 6 ovip. a 30 grados.
        
5- la mortalidad de la hembra adulta entre los 27 y los 32 dias (distribución uniforme).
6- la maduración de la pupa entre los 17 y 19 días (distribución uniforme).
7- la efectividad de la campaña publicitaria a través del vaciado de tachos en los días de mas calor.
8- "las 4 estaciones", discretizando la curva de temperatura que se tiene para buenos aires (extraído del trabajo de Otero):
        - el día 1 es 1/07 en este programa (<T>=18°C durante los 1ros 80 dias)
9- el descacharrado se hace solo en dic, enero y febrero (día 150 al 240)
10- la hibernación de los huevos en invierno.
11- la mortalidad de las hembras adultas entre 28 y 30 dias (distribución uniforme).
12- la saturación de los tachos, es decir, un número maximo de huevos permitidos por tacho
13- la transferencia de tacho, es decir, cuando un tacho satura, la hembra busca otro tacho para depositar sus huevos.

Las condiciones iniciales para cada agente mosquita tiene cuatro propiedades
    -1 o 0 (si está viva o muerta)
    -edad (avanza de a 1 dia)
    -cohorte (tacho en el que vive)
    -tiempo en el que se vuelve adulta
    -dias que va a vivir

//*************************************************************************************************************
//                                  Programa CUDA/C de mosquitas para N manzanas version (2021) 
//                                  Autoras Ana A. Gramajo y Karina Laneri
//*************************************************************************************************************
Se extiende el código serializado de Fabiana, a uno paralelizado ya que se agrega 

    - la manzana donde se encuentra el cohorte en el que vive la mosquita a las condiciones iniciales.
    - la espacialidad, considerando que mosquita puede cambiar de manzana a 1ros vecinos para depositar sus huevos cuando se satura su cohorte original.
    
Además, en esta nueva versión del código se puede elegir la distribución  de los cohortes por manzana:
    - distribución uniforme
    - distribución de Poisson.    
    
Los parámetros del código se ingresan a través del archivo parametros.h    
---------------------------------------------------------------------
*/

#include <Random123/philox.h> // philox headers
#include <Random123/u01.h>    // to get uniform deviates [0,1]
typedef r123::Philox2x32 RNG; // particular counter-based RNG


#include <random>

#include <cstdlib>
#include <iostream>
#include <fstream>
#include <vector>
#include "ran2.h"
#include <cmath>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "gpu_timer.h"
#include "parametros.h"

#include<stdio.h>


int tiempo_entre_oviposiciones(int dia){
	int t;
	//defino tiempos de oviposición y maduración en función de la temperatura	
	if(dia >= 1 && dia < 80) t=TOVIP3;      //<T>=18 
	if(dia >= 80 && dia <= 140)t=TOVIP2b;   //<T>=23
  	if(dia > 140 || dia < 260) t=TOVIP1;    //<T>=30
 	if(dia >= 260 && dia <= 320)t=TOVIP2a;  //<T>=27
 	if(dia > 320) t=TOVIP3;                 //<T>=18

	return t;}

__global__ void kernel_reproducir(int *estado, int *edad, int *tacho,int *TdV, int *pupacion, int *manzana, int *N_mobil, int dia, int tovip, int *nacidos)
{                                   

    	int indice=N_mobil[0];
	    int id = blockIdx.x*blockDim.x + threadIdx.x;
		/*Si la mosquita esta viva, esta en edad adulta, en tiempo de oviposicion y vive en un tacho disponible entonces*/
  		if(id < indice && edad[id] > pupacion[id] && edad[id]%tovip == 0) 

  		{
			RNG philox;         
			RNG::ctr_type c={{}};
			RNG::key_type k={{}};
			RNG::ctr_type r;
			k[0]=id; 
			c[1]=dia;
			c[0]=SEMILLAGLOBAL; 
			
			r = philox(c, k); 
			double azar=(u01_closed_open_32_53(r[0]));//numero aleatorios entre [0,1)
        
			// estado[id] == ESTADOVIVO && edad[id]%tovip == 0){
        
			/*Si el tacho en el que nacio tiene lugar entonces*/
				/*Antes estaba asi y andaba...*/
			int tach=tacho[id];     //tach es un entero que me indica el numero de tacho en el que esta cada mosquita
				

			int iovip=10+ (azar*25); //iovip es el numero de huevos que pone cada mosquita
						
			atomicAdd(nacidos+tach,iovip); /*sumo iovip HUEVOS en la posicion del vector nacidos (que tiene NTACHOS elementos)
				nacidos[0+tach] en el puntero al primer elemento del vector nacidos desplazado en tach elementos
				el vector nacidos tiene el numero de nacidos en cada tacho Ej: nacidos[0]=numero de nacidos en el tacho 1*/
				

  		}//cierro loop de hilos de mosquitas VIVAS   
};

//mortalidades varias	
__global__ void matar_kernel(int *estado, int *edad, int *tacho,int *pupacion, int *TdV,int *N_mobil, int dia)
{
	int N=N_mobil[0];
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	
	if(id<N){ 	
	 	RNG philox;         
	    RNG::ctr_type c={{}};
	    RNG::key_type k={{}};
	    RNG::ctr_type r;
	    k[0]=id; 
	    c[1]=dia;
	    c[0]=SEMILLAGLOBAL; 
		
    	r = philox(c, k); 
    	double azar;

     	azar=(u01_closed_open_32_53(r[0]));//numero aleatorios entre [0,1)
     	if (edad[id] < pupacion[id]){if(azar < MORACU)estado[id]=ESTADOMUERTO;}

     	azar=(u01_closed_open_32_53(r[0]));
     	if (edad[id] == pupacion[id]){if(azar < MORPUPAD)estado[id]=ESTADOMUERTO;}

     	azar=(u01_closed_open_32_53(r[0]));
		if (edad[id] > pupacion[id]){if(azar < MORAD)estado[id]=ESTADOMUERTO;}     	

        //matar viejos
		if (edad[id] >= TdV[id]){estado[id]=ESTADOMUERTO;}
	}
};

__global__ void descacharrado_kernel(int *estado, int *edad, int *tacho, int *pupacion,int *N_mobil,int dia,int ntach)
{
    	int N=N_mobil[0];
	    int id = blockIdx.x*blockDim.x + threadIdx.x;

	    if(id<N){// esta condicion es igual a decir que la mosquita estรก viva
	        if (edad[id] < pupacion[id] && tacho[id] == ntach){
	        estado[id]=ESTADOMUERTO; // se mata la mosquita
	        }
    	}
};

//elimine el estado[id]=ESTADOVIVO, ya que al final sólo quedan las mosquitas vivas

__global__ void envejecer_kernel(int *estado, int *edad,int *pupacion,int *N_mobil,int dia)
{
    	int N=N_mobil[0];
	    int id = blockIdx.x*blockDim.x + threadIdx.x;

        if(id<N){
            if(dia < 80 || dia > 320){
                if(edad[id] > pupacion[id])edad[id]++;} //ADULTAS
  		    else{
  		        edad[id]++;}
    	}
};


__global__ void delay_kernel(int *N_mobil,int *Tau,int dia)
{
    	int N=N_mobil[0];
	    int id = blockIdx.x*blockDim.x + threadIdx.x;

        if(id<N){
            if(Tau[id] > 0)Tau[id]=Tau[id] - 1; //si Tau[id]=nTau=delay, entonces Tau[id]=nTau -1
    	}
};

// functorcito para transferencia de tacho

struct transferirdetacho{
	int m;
	int tpupad;
	int* ptr;
	int cuantos;
	int dia;

	transferirdetacho(int m_,int tpupad_,int *ptr_,int cuantos_, int dia_):
	m(m_),tpupad(tpupad_),ptr(ptr_),cuantos(cuantos_), dia(dia_){};

	__device__ 
	int operator()(thrust::tuple<int,int> tup, int counter){
		int tachoactual=thrust::get<0>(tup);
		int edad=thrust::get<1>(tup);


		int tachonuevo=tachoactual;

		if(tachoactual==m && edad>=tpupad)
		{
			/* sortear nuevo tacho de la misma manzana*/
			RNG philox;         
			RNG::ctr_type c={{}};
			RNG::key_type k={{}};
			RNG::ctr_type r;
			k[0]=counter; 
			c[1]=dia;
			c[0]=SEMILLAGLOBAL; 

			r = philox(c, k); 

			float azar=(u01_closed_closed_32_53(r[0]));//generador de números aleatorios en el device entre [0,1]

			int indicedetachoelegido=int(azar*cuantos);

			
			//tachonuevo=tachoactual; // este es solo para test


			if(indicedetachoelegido<cuantos){
			tachonuevo=ptr[indicedetachoelegido];
			}

		}
		return tachonuevo;
	}
};


// functorcito  para generar randoms uniformes

// NUEVO: otro functorcito usado para las estadisticas desagregadas

// struct para generar randoms uniformes

struct uniformRanInt{
	int dia;
	int medio;
	int ancho;
	uniformRanInt(int medio_, int ancho_, int dia_):medio(medio_),ancho(ancho_),dia(dia_)
	{};

	__device__ int operator()(int i)
	{
	 	RNG philox;         
	    RNG::ctr_type c={{}};
	    RNG::key_type k={{}};
	    RNG::ctr_type r;
	    k[0]=i; 
	    c[1]=dia;
	    c[0]=SEMILLAGLOBAL; 

		r = philox(c, k); 
     	float azar=(u01_closed_open_32_53(r[0]));//numero aleatorios entre [0,1)
		return int(medio+ancho*azar);
	}
};	

// otro functorcito usado para las estadisticas desagregadas
struct acuaticoeneltacho{
	int m;
    int t;
	acuaticoeneltacho(int m_, int t_):m(m_),t(t_){};
    
	__device__ bool operator()(thrust::tuple<int,int> tupla)
	{
        int tach=thrust::get<0>(tupla);
        int edad=thrust::get<1>(tupla);
		return (tach==m && edad<t);
	}
};

struct aereaeneltacho
{
	int m;
    int t;
	aereaeneltacho(int m_, int t_):m(m_),t(t_){};   
	__device__ bool operator()(thrust::tuple<int,int> tupla)
	{
        int tach=thrust::get<0>(tupla);
        int edad=thrust::get<1>(tupla);
		return (tach==m && edad>t);
	}
};


struct acuaticoeneltachoANA{
	int m;
    int t;
	aereaeneltacho(int m_, int t_):m(m_),t(t_){};   
	__device__ bool operator()(thrust::tuple<int,int> tupla)
	{
        int tach=thrust::get<0>(tupla);
        int edad=thrust::get<1>(tupla);
		return (tach==m && edad>t);
	}
};



//functorcito para contar adultos en la población
struct poblacion_1{
	__device__ bool operator()(thrust::tuple<int,int> tupla)
	{
        int edad=thrust::get<0>(tupla);//NUEVO: cambie el orden de la tupla
        int pupacion=thrust::get<1>(tupla);
		return (edad >= pupacion);
	}
};

// functorcito para contar acuáticos en la población
struct poblacion_2{
	__device__ bool operator()(thrust::tuple<int,int> tupla)
	{
        int edad=thrust::get<0>(tupla); //NUEVO:cambie el orden de la tupla
        int pupacion=thrust::get<1>(tupla);
		return (edad < pupacion);
	}
};
/*
__global__ void tachosenmanzana(int manzananum)
{
    	int N=N_mobil[0];
	    int id = blockIdx.x*blockDim.x + threadIdx.x;

	    if(id<N){// esta condicion es igual a decir que la mosquita está viva
	    if (manzana[id] == manzananum)vectachmanzana=tacho[id];
    	}
		return (lista tachos en esa manzana);
};
*/

struct bichos{


	//defino arrays grandes en device la  info de cada mosquita. Numero_mosquitas elementos
	thrust::device_vector<int> estado;  //viva o muerta 0/1
	thrust::device_vector<int> edad;    //edad de la mosquita   
	thrust::device_vector<int> tacho;   // numero de tacho en que se encuentra cada mosquita valores=0 a NUMEROTACHOS  
	thrust::device_vector<int> TdV;     //tiempo de vida de cada mosquita
	thrust::device_vector<int> pupacion; //dia de paso de pupa a adulta de cada mosquita
	thrust::device_vector<int> manzana; //nro. de manzana de cada mosquita

	thrust::device_vector<int> Tau;     //array para almacenar la disponibilidad de los tachos. valores 0 a un tiempo dado.  


	// arrays medianos en device, numero_tachos elementos
	thrust::device_vector<int> nacidos; // tiene el num de tachos elementos, numero de nacidos por tacho


	thrust::host_vector<int> Tdispo; // NUEVO para almacenar la disponibilidad del cada tacho
	thrust::device_vector<int> d_T; //  NUEVO d_T=Tdispo


	// arrays medianos en host, numero_manzanas elementos
	std::vector<std::vector<int> > tachos_por_manzana; //tachos_por_manzana[i]=vector de tachos de manzana i 
	std::vector<std::vector<int> > disponibilidad_de_tachos_por_manzana; //NUEVO array para identificar la disponibilidad tachos por manzana 

	// numero de tachos elementos
	std::vector<int> manzana_del_tacho;
	std::vector<int> disponibilidad_del_tacho;         //NUEVO
	

	//array para almacenar nro de tachos por manzana
    thrust::device_vector<int> NroTachos;
	
	// array de un elemento = device variable
	thrust::device_vector<int> N_mobil; // Numero de bichos fluctuante (1 elemento)

	// punteros crudos a los arrays para pasarselos a kernels
	int *raw_edad;
	int *raw_tacho;
	int *raw_estado;
	int *raw_TdV;
	int *raw_pupacion;
	int *raw_N_mobil;
	int *raw_manzana;
	int *raw_nacidos;

	int *raw_Tau;
	
	//constructor	
	bichos(int N_,long *semilla){

		// alocamos el maximo posible
		estado.resize(MAXIMONUMEROBICHOS);	
		tacho.resize(MAXIMONUMEROBICHOS);	
		edad.resize(MAXIMONUMEROBICHOS);
		pupacion.resize(MAXIMONUMEROBICHOS);
		TdV.resize(MAXIMONUMEROBICHOS);
		manzana.resize(MAXIMONUMEROBICHOS);

		Tau.resize(MAXIMONUMEROBICHOS);

		N_mobil.resize(1);
		
		tachos_por_manzana.resize(NUMEROMANZANAS); 
        disponibilidad_de_tachos_por_manzana.resize(NUMEROMANZANAS); 

		manzana_del_tacho.resize(MAXIMONUMEROBICHOS);
		disponibilidad_del_tacho.resize(MAXIMONUMEROBICHOS);

		NroTachos.resize(NUMEROMANZANAS);

        Tdispo.resize(NUMEROTACHOS);//NUEVO
        d_T.resize(NUMEROTACHOS);//NUEVO

		// nacidos en cada tacho, inicialmente 0
		nacidos.resize(NUMEROTACHOS);
		thrust::fill(nacidos.begin(),nacidos.end(),0);

		thrust::fill(estado.begin(),estado.end(),0);
		thrust::fill(edad.begin(),edad.end(),0);
		thrust::fill(tacho.begin(),tacho.end(),0);
		thrust::fill(pupacion.begin(),pupacion.end(),0);
		thrust::fill(TdV.begin(),TdV.end(),0);
		thrust::fill(manzana.begin(),manzana.end(),0);


		thrust::fill(Tau.begin(),Tau.end(),0);
		
		// inicializacion raw pointers para pasarlos al kernel
		
		// inicializacion raw pointers

		raw_edad=thrust::raw_pointer_cast(edad.data());
		raw_tacho=thrust::raw_pointer_cast(tacho.data());
		raw_estado=thrust::raw_pointer_cast(estado.data());
		raw_TdV=thrust::raw_pointer_cast(TdV.data());
		raw_manzana=thrust::raw_pointer_cast(manzana.data());

    raw_Tau=thrust::raw_pointer_cast(Tau.data());

		raw_N_mobil=thrust::raw_pointer_cast(N_mobil.data());
		raw_pupacion=thrust::raw_pointer_cast(pupacion.data());
		raw_nacidos=thrust::raw_pointer_cast(nacidos.data());


        //para considerar una distribucion de Poisson de los tachos
        /*std::default_random_engine generator;
        std::poisson_distribution<int> distribution(65);

	    const int nrolls = 100000; // number of experiments
  	    const int nstars = 100;   // maximum number of stars to distribute

	    for (int i=0; i<nrolls; ++i) {
	    int number = distribution(generator);
	    if (number< N_) ++tacho[number];
	    }
	    */
	    
		//std::cout<<"VoM\ttacho\tedad\tTdV\ttpupad\tmanzana" << std::endl;

//*************************************************************************************************************
//                       condiciones iniciales para N_=NINICIAL ingresado en el archivo parametros.h
//*************************************************************************************************************
		std::cout << "******************************************************************************************" << "\n";
		std::cout << "************************  condiciones iniciales ************************************" << "\n";
		std::cout << "*******************************************************************************************" << "\n";
		std::cout << "indice i" <<"\ttachos[i] "<< "\tdispo.[i]"<<  "\tmanzana[i]  " << "\n";

		for(int i=0;i < N_;i++){
		    
    		tacho[i] = i;			                                        //tacho en el que se encuentra la mosquita

		    manzana_del_tacho[tacho[i]]=int (i/5);                          //para 5 tachos por manzana
    		//manzana_del_tacho[tacho[i]]=int(ran2(semilla)*NUMEROMANZANAS); //le asigno al tacho una manzana al azar
		    manzana[i]=manzana_del_tacho[tacho[i]];                         //manzana en la que está el tacho i
		    tachos_por_manzana[manzana[i]].push_back(tacho[i]);             //para identificar los tachos tengo en la manzana
    		
		    disponibilidad_del_tacho[tacho[i]]=0;                           //NUEVO 0 para disponible, distinto de 0 para no disponible
		    Tau[i] = disponibilidad_del_tacho[tacho[i]];                    //NUEVO disponibilidad del tacho i
    		disponibilidad_de_tachos_por_manzana[manzana[i]].push_back(Tau[i]); //NUEVO para identificar la disponibilidad de los tachos en la manzana

		    //int tachosxmanzana=tachos_por_manzana[manzana[i]].size();      //nro de tachos x manzana
			    //if(tachosxmanzana <=9){                                    //pongo hasta 9 tachos por manzana
			    estado[i] = ESTADOVIVO; 	      		                    //todas vivas inicialmente
			    edad[i] = ran2(semilla)*7+19; 	                            //todas adultas al principio 
			    pupacion[i] = TPUPAD-2+(ran2(semilla)*5);                   //dia de pupacion (entre los 15 y 19 dias)
			    TdV[i] = ran2(semilla)*6+27 ;	                            //tiempo de vida de 27 a 32
	    		std::cout << i << "\t\t" <<tacho[i] << "\t\t" << disponibilidad_del_tacho[tacho[i]] << "\t\t" << manzana[i] << "\n";

			    //}
		}

		std::cout << "*****************************************************************************************" << "\n";
		std::cout << "verificacion del llenado de tachos_por_manzana y de la disponibilidad_de_tachos por_manzana"<< "\n";
		std::cout << "*****************************************************************************************" << "\n";

		int nmanzanas=tachos_por_manzana.size();

		for(int i=0;i<nmanzanas;i++){
			std::cout << "\n\n manzana " << i << "\ntachos " << "disponibilidad " << "\n";
			int ntachos=tachos_por_manzana[i].size();   //nro de tachos por manzana
			int contTach=0;                             //contador para el nro de tachos por manzana

			for(int j=0;j<ntachos;j++){
				std::cout << (tachos_por_manzana[i])[j] << "\t " << (disponibilidad_de_tachos_por_manzana[i])[j] << "\n";
				contTach++;
				NroTachos[i]=contTach;  
			}
			std::cout << "\nNro de tachos en la manzana\t" << contTach << "\n";
			std::cout << std::endl;
			std::cout <<"----------------------------------------------------------------------------------------"<< "\n";			}
	
		std::cout << "inicializacion lista" << std::endl;
		N_mobil[0]=N_;
	};	

	// devuelve un numero de tacho random de la manzana m
	int nuevo_tacho_misma_manzana(int m){
		int ntachos=tachos_por_manzana[m].size();
		int r=int((rand()*1.0/RAND_MAX)*ntachos);		
		int nuevo_tacho=(tachos_por_manzana[m])[r];
		return nuevo_tacho;
	}
	
	// devuelve numero de manzana sorteada entre las cuatro manzanas vecinas de la manzana m
		int sorteo_manzana_vecina(int manz){
		// numero de las manzanas vecinas de una manzana
		std::vector<int> manzanas_vecinas(5);
				
		int x=manz%LADO;
		int y=int(manz/LADO);
		manzanas_vecinas[0]=(x-1+LADO)%LADO+LADO*y; //izqu
		manzanas_vecinas[1]=(x+1+LADO)%LADO+LADO*y; //derecha
		manzanas_vecinas[2]=LADO*((y-1+LADO)%LADO)+x; //abajo
		manzanas_vecinas[3]=LADO*((y+1+LADO)%LADO)+x; //arriba
		manzanas_vecinas[4]=manz; //centro
		
		int nvecinos=6; //cuento los vecinos y el centro
		int r=int((rand()*1.0/RAND_MAX)*nvecinos); //numero aleatorio entre 0 y 6
		//si sale 5 o 6 elijo el centro (es una forma trucha de darle mas probabilidad al centro)
		int manzana_sorteada;
		if(r>3)
			{
			manzana_sorteada=manz;
			}
			else{		
			manzana_sorteada=manzanas_vecinas[r];
		}
		return manzana_sorteada;
	}


	void mortalidades(int dia){

	int N=N_mobil[0];
	//mortalidades varias y muerte por vejez
	    matar_kernel<<<(N+256-1)/256,256>>>(raw_estado,raw_edad,raw_tacho,raw_pupacion,raw_TdV,raw_N_mobil, dia);	
		hipDeviceSynchronize();
	};

	void descacharrado(int dia,float *E,long *semilla){
	int N=N_mobil[0];
	
		int nmanzanas=tachos_por_manzana.size();                           //nro de manzanas
		
    	//int azar=1 + ran2(semilla)*14;                                    //nro al azar entre [1,14]
		//if(dia%azar == 0 && dia > 120 && dia < 320){          //para un vaciado de tachos entre 1 y 13 días
  		if(dia%7 == 0 && dia > 120 && dia < 320){               //para un vaciado de tachos cada 7 días

   		    for(int i=0;i < nmanzanas;i++){
   		    int NroDescach=round(NroTachos[i]*E[i]);            //nro de tachos que se van a vaciar por manzana
   		    int ntachos=tachos_por_manzana[i].size();           //nro de tachos en cada manzana i

  		    //chequeo
		    if(dia==140){
		    std::cout << "\n ----- Descacharrado para el dia 140 -----" <<"\n";
		    std::cout << "manzana: " << i << " numero de tachos en la mazanana: " << ntachos <<"\n";
		    std::cout << "indice sorteado |" << " tacho que se vacia "<<"\t|disponibilidad"<< "\n";} 

   			    for(int itach=0;itach < NroDescach;itach++){
   			        int n=ran2(semilla)*ntachos;             //(NUEVO) indice del tacho que se va a eliminar al azar
   			        //int n=itach;                           // se descacharran los primeros tachos

   		 	    	int ntach=(tachos_por_manzana[i])[n];   //tacho que se elimina 
   		 	    	int nTau= 10;// + ran2(semilla)*30;     //(NUEVO) delay para la disponibilidad del tacho
   		 	    	(disponibilidad_de_tachos_por_manzana[i])[n] =nTau; //(NUEVO) el tacho que se elimina tiene un delay de nTau días para volver a estar disponible
                    //chequeo
                    if(dia==140){std::cout << "\t" << n << "\t|\t" << ntach << " \t\t|\t" << (disponibilidad_de_tachos_por_manzana[i])[n]<<"\n";}
                //las mosquitas que viven en el tacho=ntach cambian su estado de VIVAS -> MUERTAS    
//Antes
//		if(dia%7 == 0 && dia > 120 && dia < 320){
//  			for(int itach=0;itach < descach;itach++){
//    			int ntach=ran2(&semilla)*NUMEROTACHOS;
    			//std::cout << ntach << "\n";
              
  				descacharrado_kernel<<<(N+256-1)/256,256>>>(raw_estado, raw_edad, raw_tacho, raw_pupacion, raw_N_mobil,dia,ntach);
  				hipDeviceSynchronize();
  				
			    }//cierro for para eliminar los tachos
   		    }//cierro for para las manzanas
   		}//cierro if
   		
   		//(NUEVO) Defino un array Tdisp(NUMEROTACHOS) en el host para almacenar el estado de cada tacho: disponible=0, no disponible=nTau días 
   		int cont=0;
   		
		for(int i=0;i<nmanzanas;i++){
			int ntachos=tachos_por_manzana[i].size();
            //chequeo
			if(dia==140){std::cout << "manzana: "<< i << "\n";}
			
			for(int j=0;j<ntachos;j++){
			    Tdispo[cont]=(disponibilidad_de_tachos_por_manzana[i])[j];
			    //chequeo
			        if(dia==140){std::cout << Tdispo[cont] << "\n";}
			    cont++;
			}//cierro for para tachos
		}//cierro for para manzanas
   	};

    //nacimientos
	void reproducir(int dia,int tovip)
	{
	    
		int indice=N_mobil[0];
		if(indice==0) {
			std::cout << "NO HAY MAS MOSQUITAS PARA REPRODUCIRSE" << std::endl; 	

		//exit(1);//comenté esta linea porque terminaba el programa y no era necesario
		}else{

		//nacimientos
		//antes de reproducir reinicializo en cero los nacidos en el paso anterior que ahora ya no son mas nacidos porque crecieron 
		thrust::fill(nacidos.begin(),nacidos.end(),0);

		// reproduce primero y luego ve si los tachos no están saturados para poner los huevos
		kernel_reproducir<<<(indice+256-1)/256,256>>>(raw_estado,raw_edad,raw_tacho,raw_TdV,raw_pupacion,raw_manzana,raw_N_mobil,dia,tovip,raw_nacidos);
		hipDeviceSynchronize();

			// despues de reproducir agrego todos los nacidos al final del array original, tacho a tacho
			int index=indice;
			
			for(int m=0;m<NUMEROTACHOS;m++){


				//calculo el nunmero de acuaticos en cada tacho
				int antiguos=thrust::count_if(
					thrust::make_zip_iterator(thrust::make_tuple(tacho.begin(),edad.begin())),
					thrust::make_zip_iterator(thrust::make_tuple(tacho.begin()+indice,edad.begin()+indice)),
					acuaticoeneltacho(m,TPUPAD)
				);


				//los nuevos vienen del kernel reproducir  
				int nuevos=nacidos[m];

                //(NUEVO) copio el array donde almaceno la disponibilidad de los tachos después del descacharrado que está en el host y lo llevo al device
                thrust::device_vector<int> d_T = Tdispo;

                int dispo=d_T[m]; //(NUEVO) disponibilidad por tacho m,dispo= 0 para disponible y dispo=nTau para no disponible
 
                //chequeo para un día determinado
			     //   if(dia==140){
			     //       std::cout << "m: "<< m << "\t" << dispo << "\n";//fuciona 
			     //   }
			        
				    //Ahora bien, si con los nuevos supero el maximo de huevos por tacho (SAT)y (NUEVO) el tacho está disponible
				    if(nuevos+antiguos>SAT && dispo==0){
				        
				    nuevos=SAT-antiguos;	//ponen lo que pueden en el mismo tacho
					
				    /*Para transferir de tacho*/
				    /*Muevo LOS ADULTOS a otro tacho de la misma manzana o de una manzana vecina*/
					int estamanzana = manzana_del_tacho[m];
					int manzanadeltacho = sorteo_manzana_vecina(estamanzana); //pone en la misma manzana o en una vecina
					int cuantos=(tachos_por_manzana[manzanadeltacho]).size(); //nro de tachos por manzana

					int* ptr_h=(tachos_por_manzana[manzanadeltacho]).data();

					thrust::device_vector<int> tachosDeLaManzana(cuantos);

				    	for(int k=0;k<cuantos;k++){
						tachosDeLaManzana[k]=ptr_h[k];
					    }	
					    
					int* ptr_d=thrust::raw_pointer_cast(tachosDeLaManzana.data());

					thrust::transform(
						thrust::make_zip_iterator(thrust::make_tuple(tacho.begin(),edad.begin())),
						thrust::make_zip_iterator(thrust::make_tuple(tacho.begin()+indice,edad.begin()+indice)),
						thrust::make_counting_iterator(0),
						tacho.begin(),
						transferirdetacho(m,TPUPAD,ptr_d,cuantos, dia)
					);
                    
                    dispo=d_T[m]; //NUEVO disponibilidad del nuevo tacho m luego de hacer la transferencia
                    
					//Una vez que se llenaron los tachos de la manzana, pone en las manzanas vecinas.
				    }//cierro if para transferencia de tacho
				    
				/*HASTA ACÁ MAYOR PROBABILIDAD DE TRANSFERENCIA DE TACHO EN LA MISMA MANZANA Y MENOR PORB. DE TRANSFERENCIA DE MANZANA y TACHO */
                if(dispo==0){ //NUEVO si el nuevo tacho está disponible, entonces que agregue al final de los arrays las nuevas mosquitas

				thrust::fill(estado.begin()+index,estado.begin()+index+nuevos,ESTADOVIVO);	//nacen todas vivas       
				thrust::fill(edad.begin()+index,edad.begin()+index+nuevos,1);		        //nacen con edad(dias)      
				thrust::fill(tacho.begin()+index,tacho.begin()+index+nuevos,m); 	        //nacen en el tacho m

                thrust::fill(Tau.begin()+index,Tau.begin()+index+nuevos,disponibilidad_del_tacho[m]); //(NUEVO) nacen en un tacho disponible
                
				// index en counting iteraror necesario para distintos randoms en cada tacho
				thrust::transform(
					thrust::make_counting_iterator(index),thrust::make_counting_iterator(index+nuevos),
					pupacion.begin()+index,uniformRanInt(15,5,dia)
				);
			
				
				thrust::transform(
					thrust::make_counting_iterator(index),thrust::make_counting_iterator(index+nuevos),
					TdV.begin()+index,uniformRanInt(27,6,dia)
				);

				thrust::fill(manzana.begin()+index,manzana.begin()+index+nuevos,manzana_del_tacho[m]);        
				index+=nuevos;		//actualizo el indice para me marque siempre en la ultima mosquita que nacio    

                }//cierro if linea 635

			}//cierro for para los tachos
		
		    // actualiza el indice movil hasta el ultimo bicho vivo

			if(index<MAXIMONUMEROBICHOS) {
				N_mobil[0]=index;
			}
			else{ ////satura la memoria reservada salgo del prog
				std::cout << "Demasiados Bichos!" << std::endl;
				exit(1);
			}	
				
		}//cierro else linea 523	
		
	};
	
    //Recalcular -> eliminar muertos y dejar vivos
    void recalcularN(){

		auto zip_iterator=
		thrust::make_zip_iterator(thrust::make_tuple(edad.begin(),tacho.begin(),pupacion.begin(),TdV.begin(),manzana.begin(),Tau.begin()));
		// ordenamos segun estado 0-vivo, 1-muerto
		int N=N_mobil[0];
		thrust::sort_by_key(estado.begin(), estado.begin() + N,zip_iterator);		
	
		// y ahora determinamos la posicion del primer muerto = N_mobil
		auto iter=thrust::find(estado.begin(),estado.begin() + N, ESTADOMUERTO);
		N_mobil[0]= iter-estado.begin();//me da la longitud del vector
		//std::cout << "N_mobil " << N_mobil[0] <<std::endl;
	};
	
	//población total adultos + acuáticos
	int vivos(int dia){

	int N=N_mobil[0];

    int poblacion = thrust::count(estado.begin(), estado.begin() + N, ESTADOVIVO);
	return poblacion;
	};

    //población de acuáticos
	int acuaticos(int dia){

	int N=N_mobil[0];
	
    int ac= thrust::count_if(
                thrust::make_zip_iterator(thrust::make_tuple(edad.begin(),pupacion.begin())),
                thrust::make_zip_iterator(thrust::make_tuple(edad.begin() +  N,pupacion.begin() + N)),
                poblacion_2()
            );

		return ac;
	};

    //población de adultos
	int adultos(int dia){

	int N=N_mobil[0];
	//el predicado poblacion_1()corresponde a adultos
	    int ad=thrust::count_if(
                thrust::make_zip_iterator(thrust::make_tuple(edad.begin(),pupacion.begin())),
                thrust::make_zip_iterator(thrust::make_tuple(edad.begin() +  N,pupacion.begin() + N)),
                poblacion_1()
            );

		return ad;
	};	

	//envejecer población
   	void envejecer(int dia){
	int N=N_mobil[0];
        envejecer_kernel<<<(N + 256-1)/256,256>>>(raw_estado,raw_edad,raw_pupacion,raw_N_mobil,dia);
        hipDeviceSynchronize();
	}; 
	
	//(NUEVO) disminuir el delay=nTau en 1 día para que el tacho eliminado vuelva a estar disponible
   	void delay(int dia){
	int N=N_mobil[0];

        //chequeo
		if(dia==140){std::cout << "al final del dia 140 disminuyo en 1 el delay para los tachos" << "\n";}
		
		int nmanzanas=tachos_por_manzana.size();
		for(int i=0;i<nmanzanas;i++){
			int ntachos=tachos_por_manzana[i].size();
			for(int j=0;j<ntachos;j++){
			    if((disponibilidad_de_tachos_por_manzana[i])[j]>0){(disponibilidad_de_tachos_por_manzana[i])[j]--;}
			    //chequeo
			    if(dia==140){std::cout << (disponibilidad_de_tachos_por_manzana[i])[j] << "\n"; }
			}//cierro for para disponibidad de tachos
		}//cierro for para manzanas	
	}; 
	
};

int main(){

	FILE* archivo=NULL;
	char miarch[50];
	
    //alocamos memoria para el vector que almacena la efectividad por manzana
    float *E;//array cuyos elementos esla efectividad de propaganda en cada manzana
    E= (float *)malloc((NUMEROMANZANAS)*sizeof(float));    

    //para un descacharrado fijo en cada manzana
    for(int j=0;j<NUMEROMANZANAS;j++){E[j]=PROP;}

    //alocamos memoria para los vectores donde almaceno el nro de mosquitas por dia, y la suma de todas las poblaciones
    int *Poblacion;
    Poblacion= (int *)malloc((NDIAS+1)*sizeof(int));
    for(int i=1;i<=NDIAS;i++){Poblacion[i]=0;}

    //loop para el número de corridas con distinta semilla
    for(int seed=0;seed<NITERACIONES;seed++){
    std::cout << "nro de realizacion: "<< seed+1 << "\n";
    //incializamos semilla
    //long semilla=(long )time(NULL);
    long semilla = -739;

    //para un descacharrado distinto en casa manzana, lo pongo dentro del loop para que varíe con la semilla
        //for(int j=0;j<NUMEROMANZANAS;j++){E[j]=0.4 + ran2(&semilla)*0.5;}
    
    gpu_timer Reloj_GPU;
    Reloj_GPU.tic();
    
    //inicializo
    bichos mosquitas(NINICIAL,&semilla);

    double treprod=0;
    double trecalc=0;
    double tdescacha=0;
    
        //calculo la población en cada iteración
	    for(int dia = 1; dia <= NDIAS; dia++){
	    //std::cout << "DIA" << dia << std::endl;
        int tovip=tiempo_entre_oviposiciones(dia);
	
	    //std::cout << "matar" << std::endl;
	    mosquitas.mortalidades(dia);
	    
	    gpu_timer Reloj_descacharrar;
	    Reloj_descacharrar.tic();
	    //std::cout << "descacharrar" << std::endl;
	    mosquitas.descacharrado(dia,E,&semilla); 
	    tdescacha= tdescacha+Reloj_descacharrar.tac()/60000; //de milisegundos -> minutos

	    gpu_timer Reloj_reproducir;
	    Reloj_reproducir.tic();
	    //std::cout << "reproducir" << std::endl;
	    mosquitas.reproducir(dia,tovip);
	    treprod= treprod+Reloj_reproducir.tac()/60000; //de milisegundos -> minutos
   

	    gpu_timer Reloj_recalcular;
	    Reloj_recalcular.tic();
	    //std::cout << "recalcular indice de mosquitas vivas" << std::endl;
	    mosquitas.recalcularN(); 
	    trecalc= trecalc+Reloj_recalcular.tac()/60000; //de milisegundos -> minutos
	
        //en esta versión del programa solo considero hembras
	    int vivas=mosquitas.vivos(dia);
	    int adultas=mosquitas.adultos(dia);
	    int acuaticas=mosquitas.acuaticos(dia);

	    //std::cout << "envejecer poblacion" << std::endl;
	    mosquitas.envejecer(dia);
	    mosquitas.delay(dia); //(NUEVO) disminuyo en un dia el delay=nTau de los tachos que fueron vaciados
	    Poblacion[dia]= vivas;//guardo en un vector el nro de mosquitas para una determinada semilla
	    }//cierro loop para dias

       //tiempos de cáculo en GPU***************************************************		
       double t=Reloj_GPU.tac()/60000; //de milisegundos -> minutos
       printf("Tiempo de cálculo total para Población total de mosquitas en GPU: %lf minutos\n",t);
       printf("Tiempo en descacharrar: %lf minutos\n",tdescacha);
       printf("Tiempo en reproducir: %lf minutos\n",treprod);
       printf("Tiempo en recalcular: %lf minutos\n",trecalc);	
       //****************************************************************************
       std::cout << "\n";
        //miarch es un string de caracteres donde guardo el nombre del archivo cambiando la semilla con cada iteracion
    
	    sprintf(miarch,"POBLACION_%d.txt",seed);
    	archivo=fopen(miarch,"w");
    	    for (int i=1;i<=NDIAS;i++){
		    fprintf(archivo,"%d\t%d \n",i,Poblacion[i]);}
		fclose(archivo); 
    }//cierro loop para el número de ITERACIONES
    
return 0;		


}// end for main
