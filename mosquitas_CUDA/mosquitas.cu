#include "hip/hip_runtime.h"
#include <philox.h> // philox headers
#include <u01.h>    // to get uniform deviates [0,1]
typedef r123::Philox2x32 RNG; // particular counter-based RNG

#include <hiprand.h>
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <vector>
#include "ran2.h"
#include <cmath>
#include <thrust/device_vector.h>
#include "gpu_timer.h"
#include "parametros.h"

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)
    
std::ofstream outfile, outfile1, outfile2,outfile3,outfile4,outfile5;
	
int tiempo_entre_oviposiciones(int dia){
	int t;
	//defino tiempos de oviposición y maduración en función de la temperatura	
	if(dia >= 1 && dia < 80) t=TOVIP3;      //<T>=18 // acá es necesario definirlo tomando el extremo
	if(dia >= 80 && dia <= 140)t=TOVIP2b;   //<T>=23
  	if(dia > 140 || dia < 260) t=TOVIP1;    //<T>=30
 	if(dia >= 260 && dia <= 320)t=TOVIP2a;  //<T>=27
 	if(dia > 320) t=TOVIP3;                 //<T>=18

	return t;}

__global__ void kernel_reproducir(int *estado, int *edad, int *tacho,int *TdV, int *pupacion,int *manzana, int *N_mobil, int dia, int tovip, int *nacidos)
{                                   

    	int indice=N_mobil[0];
	    int id = blockIdx.x*blockDim.x + threadIdx.x;
		/*Si la mosquita esta viva, esta en edad adulta y en el tiempo de oviposicion entonces*/
  		if(id < indice && edad[id] > pupacion[id] && edad[id]%tovip == 0) //calculo todo solo sobre las vivas 
  		{
			RNG philox;         
			RNG::ctr_type c={{}};
			RNG::key_type k={{}};
			RNG::ctr_type r;
			k[0]=id; 
			c[1]=dia;
			c[0]=SEMILLAGLOBAL; 
			
			r = philox(c, k); 
			double azar=(u01_closed_open_32_53(r[0]));//numero aleatorios entre [0,1)

			// estado[id] == ESTADOVIVO && edad[id]%tovip == 0){
			/*Si el tacho en el que nacio tiene lugar entonces*/
				/*Antes estaba asi y andaba...*/
			int tach=tacho[id];     //tach es un entero que me indica el numero de tacho en el que esta cada mosquita
				

			int iovip=10+ (azar*25); //iovip es el numero de huevos que pone cada mosquita
						
			atomicAdd(nacidos+tach,iovip); /*sumo iovip HUEVOS en la posicion del vector nacidos (que tiene NTACHOS elementos)
				nacidos[0+tach] en el puntero al primer elemento del vector nacidos desplazado en tach elementos
				el vector nacidos tiene el numero de nacidos en cada tacho Ej: nacidos[0]=numero de nacidos en el tacho 1*/
				
				
		    //cierro loop para mosquitas maduras
  		}//cierro loop de hilos de mosquitas VIVAS   
};

//mortalidades varias	
__global__ void matar_kernel(int *estado, int *edad, int *tacho,int *pupacion, int *TdV,int *N_mobil, int dia)
{
	int N=N_mobil[0];
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	
	if(id<N){	// es lo mismo que estado[id]==ESTADOVIVO && 	
	 	RNG philox;         
	    RNG::ctr_type c={{}};
	    RNG::key_type k={{}};
	    RNG::ctr_type r;
	    k[0]=id; 
	    c[1]=dia;
	    c[0]=SEMILLAGLOBAL; 
		
    	r = philox(c, k); 
    	double azar;
    	/*CORRECCIÓN: ahora sorteo tres nros al azar y  comparo con las mortalidades,en coherencia a lo que hacía la función en SERIAL*/
    	
     	azar=(u01_closed_open_32_53(r[0]));//numero aleatorios entre [0,1)
     	if (edad[id] < pupacion[id]){if(azar < MORACU)estado[id]=ESTADOMUERTO;}

     	azar=(u01_closed_open_32_53(r[0]));
     	if (edad[id] == pupacion[id]){if(azar < MORPUPAD)estado[id]=ESTADOMUERTO;}

     	azar=(u01_closed_open_32_53(r[0]));
		if (edad[id] > pupacion[id]){if(azar < MORAD)estado[id]=ESTADOMUERTO;}     	

        //envejecer
		if (edad[id] >= TdV[id]){estado[id]=ESTADOMUERTO;}
	}
};

__global__ void descacharrado_kernel(int *estado, int *edad, int *tacho, int *pupacion,int *N_mobil,int dia, int ntach)
{
    	int N=N_mobil[0];
	    int id = blockIdx.x*blockDim.x + threadIdx.x;

	    if(id<N){// esta condicion es igual a decir que la mosquita estรก viva
	    if (edad[id] < pupacion[id] && tacho[id] == ntach)estado[id]=ESTADOMUERTO;
    	}
};

//elimine el estado[id]=ESTADOVIVO, ya que al final sólo quedan las mosquitas vivas
__global__ void envejecer_kernel(int *estado, int *edad,int *pupacion,int *N_mobil,int dia)
{
    	int N=N_mobil[0];
	    int id = blockIdx.x*blockDim.x + threadIdx.x;

        if(id<N){
            if(dia < 80 || dia > 320){
                if(edad[id] > pupacion[id])edad[id]++;} //ADULTAS
  		    else{
  		        edad[id]++;}
    	}
};


struct transferirdetacho{
	int m;
	int tpupad;
	int* ptr;
	int cuantos;
	int dia;

	transferirdetacho(int m_,int tpupad_,int *ptr_,int cuantos_, int dia_):
	m(m_),tpupad(tpupad_),ptr(ptr_),cuantos(cuantos_), dia(dia_){};

	__device__ 
	int operator()(thrust::tuple<int,int> tup, int counter){
		int tachoactual=thrust::get<0>(tup);
		int edad=thrust::get<1>(tup);
		 
		int tachonuevo=tachoactual; //me parece que esto debería ser sólo "int  tachonuevo";
		if(tachoactual==m && edad>=tpupad)
		{
			/* sortear nuevo tacho de la misma manzana*/
			RNG philox;         
			RNG::ctr_type c={{}};
			RNG::key_type k={{}};
			RNG::ctr_type r;
			k[0]=counter; 
			c[1]=dia;
			c[0]=SEMILLAGLOBAL; 

			r = philox(c, k); 
			float azar=(u01_closed_closed_32_53(r[0])); //distribución uniforme de nro randoms entre [0,1]
			int indicedetachoelegido=int(azar*cuantos);//cuantos: nro de tachos en la manzana

			
			//tachonuevo=tachoactual; // este es solo para test

			if(indicedetachoelegido<cuantos)// agregar acá que tiene indicedeltachonuevo != m
			tachonuevo=ptr[indicedetachoelegido];
		}
		return tachonuevo;
	}
};

// struct para generar randoms uniformes
struct uniformRanInt{
	int dia;
	int medio;
	int ancho;
	uniformRanInt(int medio_, int ancho_, int dia_):medio(medio_),ancho(ancho_),dia(dia_)
	{};

	__device__ int operator()(int i)
	{
	 	RNG philox;         
	    RNG::ctr_type c={{}};
	    RNG::key_type k={{}};
	    RNG::ctr_type r;
	    k[0]=i; 
	    c[1]=dia;
	    c[0]=SEMILLAGLOBAL; 

		r = philox(c, k); 
     	float azar=(u01_closed_open_32_53(r[0]));//numero aleatorios entre [0,1)
		return int(medio+ancho*azar);
	}
};	


// otro functorcito usado para las estadisticas desagregadas
struct acuaticoeneltacho{
	int m;
    int t;
	acuaticoeneltacho(int m_, int t_):m(m_),t(t_){};
    
	__device__ bool operator()(thrust::tuple<int,int> tupla)
	{
        int tach=thrust::get<0>(tupla);
        int edad=thrust::get<1>(tupla);
		return (tach==m && edad<t);
	}
};

struct aereaeneltacho
{
	int m;
    int t;
	aereaeneltacho(int m_, int t_):m(m_),t(t_){};   
	__device__ bool operator()(thrust::tuple<int,int> tupla)
	{
        int tach=thrust::get<0>(tupla);
        int edad=thrust::get<1>(tupla);
		return (tach==m && edad>t);
	}
};


struct acuaticoeneltachoANA{
	int m;

	acuaticoeneltachoANA(int m_):m(m_){};
    
	__device__ bool operator()(thrust::tuple<int,int,int> tupla)
	{
        int tacho=thrust::get<0>(tupla);
        int edad=thrust::get<1>(tupla);
        int pupacion=thrust::get<2>(tupla);
		return (tacho==m && edad < pupacion);
	}
};

//functorcito para contar adultos en la población
struct poblacion_1{
	__device__ bool operator()(thrust::tuple<int,int> tupla)
	{
        int edad=thrust::get<0>(tupla);//NUEVO: cambie el orden de la tupla
        int pupacion=thrust::get<1>(tupla);
		return (edad >= pupacion);
	}
};

// functorcito para contar acuáticos en la población
struct poblacion_2{
	__device__ bool operator()(thrust::tuple<int,int> tupla)
	{
        int edad=thrust::get<0>(tupla); //NUEVO:cambie el orden de la tupla
        int pupacion=thrust::get<1>(tupla);
		return (edad < pupacion);
	}
};

// un functorcito usado para las estadisticas desagregadas
struct iguala{
	int m;
	iguala(int m_):m(m_){};

	__device__ bool operator()(int man)
	{
		return man==m;
	}
};


/*
__global__ void tachosenmanzana(int manzananum)
{
    	int N=N_mobil[0];
	    int id = blockIdx.x*blockDim.x + threadIdx.x;
	    if(id<N){// esta condicion es igual a decir que la mosquita está viva
	    if (manzana[id] == manzananum)vectachmanzana=tacho[id];
    	}
		return (lista tachos en esa manzana);
};
*/

struct bichos{

	// arrays grandes en device, numero_mosquitas elementos: info de cada mosquita
	thrust::device_vector<int> estado; // Vivo o Muerto 0/1 
	thrust::device_vector<int> edad; // tiene num de mosqu elementos y los valores van de 0 a MAXIMAEDAD   
	thrust::device_vector<int> tacho; // numero de tacho en que se encuentra cada mosquita valores=0 a NUMEROTACHOS   
	thrust::device_vector<int> TdV;  //tiempo de vida de cada mosquita
	thrust::device_vector<int> pupacion; //dia de paso de pupa a adulta de cada mosquita
	thrust::device_vector<int> manzana; //numero de manzana de cada mosquita

	// arrays medianos em device, numero_tachos elementos
	thrust::device_vector<int> nacidos; // tiene el num de tachos elementos, numero de nacidos por tacho

	// arrays mediano en host, numero_manzanas elementos
	std::vector<std::vector<int> > tachos_por_manzana; //tachos_por_manzana[i]=vector de tachos de manzana i 

	// numero de tachos elementos
	std::vector<int> manzana_del_tacho;

    	//array para almacenar nro de tachos por manzana
   	thrust::device_vector<int> NroTachos;
    
	// array de un elemento = device variable
	thrust::device_vector<int> N_mobil; // Numero de bichos fluctuante (1 elemento)

	// punteros crudos a los arrays para pasarselos a kernels
	int *raw_edad;
	int *raw_tacho;
	int *raw_estado;
	int *raw_TdV;
	int *raw_pupacion;
	int *raw_N_mobil;
	int *raw_manzana;
	int *raw_nacidos;
	
	//constructor	
	bichos(int N_){

		// alocamos el maximo posible
		estado.resize(MAXIMONUMEROBICHOS);	
		tacho.resize(MAXIMONUMEROBICHOS);	
		edad.resize(MAXIMONUMEROBICHOS);
		pupacion.resize(MAXIMONUMEROBICHOS);
		TdV.resize(MAXIMONUMEROBICHOS);
		manzana.resize(MAXIMONUMEROBICHOS);

		N_mobil.resize(1);

		//tachos_por_manzana.resize(int(N_/5.0));
		tachos_por_manzana.resize(NUMEROMANZANAS);

		manzana_del_tacho.resize(MAXIMONUMEROBICHOS);

        NroTachos.resize(NUMEROMANZANAS);
        
		// nacidos en cada tacho, inicialmente 0
		nacidos.resize(NUMEROTACHOS);
		thrust::fill(nacidos.begin(),nacidos.end(),0);

		thrust::fill(estado.begin(),estado.end(),0);
		thrust::fill(edad.begin(),edad.end(),0);
		thrust::fill(tacho.begin(),tacho.end(),0);
		thrust::fill(pupacion.begin(),pupacion.end(),0);
		thrust::fill(TdV.begin(),TdV.end(),0);
		thrust::fill(manzana.begin(),manzana.end(),0);

		// inicializacion raw pointers
		raw_edad=thrust::raw_pointer_cast(edad.data());
		raw_tacho=thrust::raw_pointer_cast(tacho.data());
		raw_estado=thrust::raw_pointer_cast(estado.data());
		raw_TdV=thrust::raw_pointer_cast(TdV.data());
		raw_manzana=thrust::raw_pointer_cast(manzana.data());
		raw_N_mobil=thrust::raw_pointer_cast(N_mobil.data());
		raw_pupacion=thrust::raw_pointer_cast(pupacion.data());
		raw_nacidos=thrust::raw_pointer_cast(nacidos.data());
	    
        outfile3 << "CONDICIONES INICIALES" << std::endl;
		outfile3 <<"estado\tedad\ttacho\tmanzana" << std::endl;
		/*condiciones iniciales donde N sera el numero de bichos*/
		for(int i=0;i < N_;i++){
			estado[i] = ESTADOVIVO; 		             //todos vivos inicialmente
			tacho[i] = i;				                 //tacho en el que se encuentra la mosquita
			edad[i] = ran2(&semilla)*7+19; 	             //edad: son todas adultas al principio 
			pupacion[i] = TPUPAD-2+(ran2(&semilla)*5);   //dia de pupacion (entre los 15 y 19 dias)
			TdV[i] = ran2(&semilla)*6+27 ;	             //tiempo de vida de 27 a 32
			manzana[i] = NUMEROMANZANAS*ran2(&semilla);  //manzana en la que se encuentra
			//manzana[i] = (int) (tacho[i]/5);            //manzana en la que se encuentra
			tachos_por_manzana[manzana[i]].push_back(tacho[i]);
			manzana_del_tacho[tacho[i]]=manzana[i];

			outfile3 << estado[i] << "\t\t" << edad[i] << "\t\t" << tacho[i] << "\t\t" << manzana[i] << "\n";
		}

		// una verificacion del llenado de tachos_por_manzana
		int nmanzanas=tachos_por_manzana.size();
		for(int i=0;i<nmanzanas;i++){
			outfile3 << "\n\n manzana " << i << "\n tachos: ";
			int ntachos=tachos_por_manzana[i].size();
			int contTach=0;
			for(int j=0;j<ntachos;j++){
				outfile3 << (tachos_por_manzana[i])[j] << ", ";//tacho que se encuentra en la manzana
				contTach++;
				NroTachos[i]=contTach; //contador para contar nro de tachos por manzana
			}
			outfile3 << "\n Nro de tachos en la manzana\t" << contTach << "\n";
			outfile3 << std::endl;
		}
		outfile3 << "\n";
		outfile3 << "manzana del tacho 16 " << manzana_del_tacho[16] << std::endl;
		outfile3 << "manzana del tacho 10 " << manzana_del_tacho[10] << std::endl;
		outfile3 << "manzana del tacho 1 " << manzana_del_tacho[1] << std::endl;
    	outfile3 << "\n";

        outfile3 << "NTachos x manzana" << std::endl;
        for(int i=0;i<nmanzanas;i++){
        outfile3 <<  NroTachos[i] << "\n";
        }

		outfile3 << "inicializacion lista:\n" << "Nro de Tachos\t" << NUMEROTACHOS << "\n" << "Nro de Manzanas\t" << NUMEROMANZANAS <<std::endl;
		N_mobil[0]=N_;
	};	

	// devuelve un numero de tacho random de la manzana m
	int nuevo_tacho_misma_manzana(int m){
		int ntachos=tachos_por_manzana[m].size();
		int r=int((rand()*1.0/RAND_MAX)*ntachos);		
		int nuevo_tacho=(tachos_por_manzana[m])[r];
		return nuevo_tacho;
	}
	
	void mortalidades(int dia){

	int N=N_mobil[0];

	    matar_kernel<<<(N+256-1)/256,256>>>(raw_estado,raw_edad,raw_tacho,raw_pupacion,raw_TdV,raw_N_mobil, dia);	
		hipDeviceSynchronize();
	};

	//descachrarrado

	//void descacharrado(int dia,float *E){//DESCOMENTAR para una efectividad de propaganda distinta para cada manzana y cte durante los dias de descacharrado. 

	void descacharrado(int dia){//DESCOMENTAR para una efectividad de propaganda distinta para cada manzana y que varia durante los dias de descacharrado. 

	//void descacharrado(int dia,int descach){//DESCOMENTAR para una efectividad fija 0.6, igual para todas las manzanas, y constante durante los dias de descacharrado.	
		/*if(dia%7 == 0 && dia > 120 && dia < 320){
  			for(int itach=0;itach < descach;itach++){
    			int ntach=ran2(&semilla)*NUMEROTACHOS;
    			std::cout << "tacho que se descacharran\t" << ntach << "\n";
  				descacharrado_kernel<<<(N+256-1)/256,256>>>(raw_estado, raw_edad, raw_tacho, raw_pupacion, raw_N_mobil,dia,ntach);
  				hipDeviceSynchronize();
			}    	
   		}*/

                int N=N_mobil[0];

   		thrust::device_vector<int> D(28); //array para almacenar los dias que se descacharra
   		thrust::device_vector<float> E(NUMEROMANZANAS);  //un array para almacenar la efectividad de propaganda x manzana
   		
   		int j=0;
   		
   		if(dia%7 == 0 && dia > 120 && dia < 320){
   		    D[j]=dia;
   		    for(int i=0;i < NUMEROMANZANAS;i++){
   		    E[i]=ran2(&semilla)*0.8;//para generar distinta efectividad por manzana y por dia que se descacharra
   		    int NroDescach=round(NroTachos[i]*E[i]);//Nro de Tachos de esa manzana por la efectividad
   		    outfile5 << D[j] << "\t" << i << "\t" << E[i] << "\t" << NroTachos[i] << "\t" << NroDescach << std::endl;
  			    for(int itach=0;itach < NroDescach;itach++){
    	    			int ntach=(tachos_por_manzana[i])[itach];//identifica los tachos que se encuentran en la manzana para eliminar
  				descacharrado_kernel<<<(N+256-1)/256,256>>>(raw_estado, raw_edad, raw_tacho, raw_pupacion, raw_N_mobil,dia,ntach);
  				hipDeviceSynchronize();
			    }//cierro for para eliminar los tachos
   		    }//cierro for para las manzanas
   		    outfile5 << "\n";
   		    j++;//incremento el contador para los elementos en D[j]
   		}//cierro if
	};


    //nacimientos
	void reproducir(int dia,int tovip)
	{
	    
		int indice=N_mobil[0];
		if(indice==0) {
			std::cout << "NO HAY MAS MOSQUITAS PARA REPRODUCIRSE" << std::endl; 	
		exit(1);
		}else{

		//nacimientos
		//std::cout << "antes kernel reproducir " << std::endl;
		//reinicializo en cero los nacidos en el paso anterior que ahora ya no son mas nacidos porque crecieron 
		thrust::fill(nacidos.begin(),nacidos.end(),0);

		// reproduce, calculando nacidos por tacho antes
		kernel_reproducir<<<(indice+256-1)/256,256>>>(raw_estado,raw_edad,raw_tacho,raw_TdV,raw_pupacion,raw_manzana,raw_N_mobil,dia,tovip,raw_nacidos);
		hipDeviceSynchronize();

		//std::cout << "despues kernel reproducir " << std::endl; 
			// agrego todos los nacidos al final del array original, tacho a tacho
			int index=indice;
			for(int m=0;m<NUMEROTACHOS;m++){
				//calculo el nunmero de acuaticos en cada tacho
				int antiguos=thrust::count_if(
					thrust::make_zip_iterator(thrust::make_tuple(tacho.begin(),edad.begin())),
					thrust::make_zip_iterator(thrust::make_tuple(tacho.begin()+indice,edad.begin()+indice)),
					acuaticoeneltacho(m,TPUPAD)
				);

				 //NUEVO: cuando el TPUPAD es variable, indice=nro de bichos hasta el momento
                /*int antiguos=thrust::count_if(
                thrust::make_zip_iterator(thrust::make_tuple(tacho.begin(), edad.begin(),pupacion.begin())),
                thrust::make_zip_iterator(thrust::make_tuple(tacho.begin() + indice, edad.begin() + indice, pupacion.begin() + indice)),
                acuaticoeneltachoANA(m)
                );*/ 
				
				//std::cout << "Acuaticos en Tacho " << antiguos << " TACHO "<< m <<std::endl;
				//los nuevos vienen del kernel reproducir  
				int nuevos=nacidos[m];

				if(nuevos+antiguos>SAT){
					nuevos=SAT-antiguos;	
					
				/*NUEVO Para transferir de tacho*/				
					int manzanadeltacho = manzana_del_tacho[m];
					int cuantos=(tachos_por_manzana[manzanadeltacho]).size();

					std::cout << "tacho que se satura " << m <<" y la manzana del tacho saturado es " << manzanadeltacho;
					std::cout << ", en esa manzana hay " << cuantos << " tachos para sortear\n";

					
					int* ptr_h=(tachos_por_manzana[manzanadeltacho]).data();
					thrust::device_vector<int> tachosDeLaManzana(cuantos);
					for(int k=0;k<cuantos;k++){
						tachosDeLaManzana[k]=ptr_h[k];
						std::cout << "Manzana " << m << " tacho " << tachosDeLaManzana[k] << "\n";
					}					
					
					int* ptr_d=thrust::raw_pointer_cast(tachosDeLaManzana.data());
					
					/*Esta transformación aplica una función unaria a cada elemento de una secuencia de entrada y almacena el resultado en la posición correspondiente en una secuencia de salida. En este caso aplica la función unaria transferirdetacho() a tacho */					
					thrust::transform(
						thrust::make_zip_iterator(thrust::make_tuple(tacho.begin(),edad.begin())),
						thrust::make_zip_iterator(thrust::make_tuple(tacho.begin()+indice,edad.begin()+indice)),
						thrust::make_counting_iterator(0),
						tacho.begin(),
						transferirdetacho(m,TPUPAD,ptr_d,cuantos, dia)
					);
				}
				/*HASTA ACA TRANSFIERE DE TACHO*/

			    std::cout << "NUEVOS NACIDOS " << nuevos <<" TACHO "<< m <<std::endl; 

				thrust::fill(estado.begin()+index,estado.begin()+index+nuevos,ESTADOVIVO);	        //NUEVO	
				thrust::fill(edad.begin()+index,edad.begin()+index+nuevos,0);		                //NUEVO	
				thrust::fill(tacho.begin()+index,tacho.begin()+index+nuevos,m); 	                //nacen en el tacho m

				//thrust::fill(pupacion.begin()+index,pupacion.begin()+index+nuevos,15);

				// index en counting iteraror necesario para distintos randoms en cada tacho
				thrust::transform(
					thrust::make_counting_iterator(index),thrust::make_counting_iterator(index+nuevos),
					pupacion.begin()+index,uniformRanInt(15,5,dia)
				);
			
				
				thrust::transform(
					thrust::make_counting_iterator(index),thrust::make_counting_iterator(index+nuevos),
					TdV.begin()+index,uniformRanInt(27,6,dia)
				);

                // la mosquita pone huevos en el tacho m de la manzana[tacho[m]]
				thrust::fill(manzana.begin()+index,manzana.begin()+index+nuevos,manzana_del_tacho[m]);         
				index+=nuevos;		//actualizo el indice para me marque siempre en la ultima mosquita que nacio 
			}//cierro for
		
		// actualiza el indice movil hasta el ultimo bicho vivo
			if(index<MAXIMONUMEROBICHOS) {
				N_mobil[0]=index;
			}
			else{ ////satura la memoria reservada salgo del prog
				std::cout << "Demasiados Bichos!" << std::endl;
				exit(1);
			}	
				
		}	
		
	};
	
    //Recalcular -> eliminar muertos y dejar vivos
    void recalcularN(){

		auto zip_iterator=
		thrust::make_zip_iterator(thrust::make_tuple(edad.begin(),tacho.begin(),pupacion.begin(),TdV.begin(),manzana.begin()));
		// ordenamos segun estado 0-vivo, 1-muerto
		int N=N_mobil[0];
		thrust::sort_by_key(estado.begin(), estado.begin() + N,zip_iterator);		
	
		// y ahora determinamos la posicion del primer muerto = N_mobil
		auto iter=thrust::find(estado.begin(),estado.begin() + N, ESTADOMUERTO);
		N_mobil[0]= iter-estado.begin();//me da la longitud del vector
		//std::cout << "N_mobil " << N_mobil[0] <<std::endl;
	};
	
	int vivos(int dia){

	int N=N_mobil[0];

    int poblacion = thrust::count(estado.begin(), estado.begin() + N, ESTADOVIVO);
	return poblacion;
	};

    //poblaciรณn de acuรกticos
	int acuaticos(int dia){

	int N=N_mobil[0];
	
    int ac= thrust::count_if(
                thrust::make_zip_iterator(thrust::make_tuple(edad.begin(),pupacion.begin())),
                thrust::make_zip_iterator(thrust::make_tuple(edad.begin() +  N,pupacion.begin() + N)),
                poblacion_2()
            );

		return ac;
	};

    //población de adultos
	int adultos(int dia){

	int N=N_mobil[0];
	//el predicado poblacion_1()corresponde a adultos
	    int ad=thrust::count_if(
                thrust::make_zip_iterator(thrust::make_tuple(edad.begin(),pupacion.begin())),
                thrust::make_zip_iterator(thrust::make_tuple(edad.begin() +  N,pupacion.begin() + N)),
                poblacion_1()
            );

		return ad;
	};	

	//envejecer población
   	void envejecer(int dia){
	int N=N_mobil[0];
        envejecer_kernel<<<(N + 256-1)/256,256>>>(raw_estado,raw_edad,raw_pupacion,raw_N_mobil,dia);
        hipDeviceSynchronize();
	}; 
		// Estadisticas de distinto tipo sobre el array de bichos
	//void imprimir_estadisticas(int Manz,int dia){
	void imprimir_estadisticas(int dia){
		int N=N_mobil[0];
		
		std::cout << "hay " << N << " mosquitas en total" << std::endl;
        if(dia==210){
        
        outfile3 << "hay " << N << " mosquitas en total en el dia\t" << dia << std::endl;    
		//for(int i=0;i<Manz;i++){
        for(int m=0;m<NUMEROMANZANAS;m++){
            //ciento cuantas mosquitas hay en cada manzana
		    int contarMosq=thrust::count_if(manzana.begin(),manzana.begin()+N,iguala(m));
		    outfile3 << "hay\t" << contarMosq << "\tmosquitas en la manzana\t" << m << std::endl;
        }    
        outfile3 << "\n";
        
        for(int t=0;t<NUMEROTACHOS;t++){
            //cuento cuantos mosquitas hay en cada tacho
            int contarTach=thrust::count_if(tacho.begin(),tacho.begin()+N,iguala(t));
            outfile3 << "hay\t"<< contarTach << "\tmosquitas en el tacho\t" << t << "\t de la manzana\t" << manzana_del_tacho[t] << std::endl;
		}
		
		    outfile3 << "\n";
		    int nmanzanas=tachos_por_manzana.size();
		    for(int i=0;i<nmanzanas;i++){
			outfile3 << "\n\n manzana " << i << "\n tachos: ";
			int ntachos=tachos_por_manzana[i].size();
			int contTach=0;
			    for(int j=0;j<ntachos;j++){
				outfile3 << (tachos_por_manzana[i])[j] << ", ";
				contTach++;
			    }
			    
			outfile3 << "\n";
			outfile3 << " Nro de tachos en la manzana\t" << contTach;
			outfile3 << std::endl;
		    }
		
        };//CIERRO IF
		
        outfile3 << "\n";
        
        for(int m=0;m<NUMEROMANZANAS;m++){
            //ciento cuantas mosquitas hay en cada manzana
		    int contarMosq=thrust::count_if(manzana.begin(),manzana.begin()+N,iguala(m));
		    outfile4 << dia <<"\t" << m << "\t" << contarMosq << "\n";
        }
        outfile4 << "\n";
	};
};



int main(){

    outfile.open("Poblacion_total_GPU.dat");		//imprime población total de mosquitas hembras: adultas + acuáticas
    outfile1.open("Poblacion_adultos_GPU.dat");		//imprime poblacion de mosquitas hembras adultas
    outfile2.open("Poblacion_acuaticos_GPU.dat");	//imprime población de mosquitas hembras acuáticas
    outfile3.open("Condiciones_iniciales_GPU.dat");	//imprime condiciones iniciales
    outfile4.open("Dia_vs_manzana_vs_N.dat");		//imprime en columnas Dia | manzana | Nro de mosquitas en la manzana
    outfile5.open("Dia_vs_manzana_vs_efectividad.dat");	//imprime en columnas Dia | manzana | efectividad | Nro de Tachos | Nro de tachos que se descacharran
    
    outfile << "dia\t" << "N"<< std::endl;//N=Adultos + Acuáticos
    outfile1 << "dia\t" << "Ad"<< std::endl; //Adultos
    outfile2 << "dia\t" << "Ac" << std::endl;//Acuáticos
    outfile5 << "dia\t" << "manz.\t" << "efect.\t" << "NTachos\t"<< "NDescach"<<std::endl;

	//int descach=round(NUMEROTACHOS*PROP);//cantidad de tachos que vacío con la propaganda

	gpu_timer Reloj_GPU;
	Reloj_GPU.tic();
	
	/*NUMEROTACHOS=NRO DE MOSQUITAS es un valor que se ingresa en archivo parametro.h*/		
	bichos mosquitas(NUMEROTACHOS);
    
	for(int dia = 1; dia <= NDIAS; dia++){
	std::cout << "DIA" << dia << std::endl;
    	int tovip=tiempo_entre_oviposiciones(dia);
	
	std::cout << "matar" << std::endl;
	mosquitas.mortalidades(dia);//fusione muerte x vejez con mortalidades varias en un solo kernel

	std::cout << "descacharrar" << std::endl;
	//mosquitas.descacharrado(dia,descach); //DESCOMENTAR para una efectividad fija 0.6, igual para todas las manzanas, y constante durante los dias de descacharrado.
    	mosquitas.descacharrado(dia);//DESCOMENTAR para una efectividad de propaganda distinta para cada manzana y que varia durante los dias de descacharrado. 
    	//mosquitas.descacharrado(dia,E);//DESCOMENTAR para una efectividad de propaganda distinta para cada manzana y cte durante los dias de descacharrado. 

	std::cout << "reproducir" << std::endl;
	mosquitas.reproducir(dia,tovip);

	std::cout << "recalcular indice de mosquitas vivas" << std::endl;
	mosquitas.recalcularN(); 

	int vivas=mosquitas.vivos(dia);
	int adultos=mosquitas.adultos(dia);
	int acuaticos=mosquitas.acuaticos(dia);

	outfile << dia << "\t" << vivas << std::endl;
	outfile1 << dia << "\t" << adultos << std::endl;
	outfile2 << dia << "\t" << acuaticos << std::endl;

	std::cout << "envejecer poblacion" << std::endl;
	mosquitas.envejecer(dia);
	
	//mosquitas.imprimir_estadisticas(NUMEROMANZANAS,dia)
	mosquitas.imprimir_estadisticas(dia);
	std::cout << "\n";
	}
    double t=Reloj_GPU.tac()/60000; //de milisegundos -> minutos
    printf("Tiempo en GPU: %lf minutos\n",t);
//cierro archivos
outfile.close();
outfile1.close();
outfile2.close();
outfile3.close();
outfile4.close();
outfile5.close();

/*Cleanup*/
/*CURAND_CALL(hiprandDestroyGenerator(rng));
CUDA_CALL(hipFree(poisson_numbers_d));
free(poisson_numbers_h);
*/
return 0;							
}// end for main
